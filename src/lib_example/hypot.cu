#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"

using namespace std;

__global__ void hypotKernel(float* A, float* B, float* C, int len) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx > len) { return; }
  C[idx] = sqrt( A[idx]*A[idx] + B[idx]*B[idx] );
}

extern "C"
int gpuHypot(float* A, float* B, float* C, int len) {
  // pick best GPU:
  int devID = gpuGetMaxGflopsDeviceId();
  checkCudaErrors( hipSetDevice(devID) );

  // allocate and initialize GPU memory:
  float* A_G;
  float* B_G;
  float* C_G;
  checkCudaErrors( hipMalloc((float**) &A_G, sizeof(float) * len) );
  checkCudaErrors( hipMalloc((float**) &B_G, sizeof(float) * len) );
  checkCudaErrors( hipMalloc((float**) &C_G, sizeof(float) * len) );
  checkCudaErrors( hipMemcpy(A_G, A, len*sizeof(float), hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(B_G, B, len*sizeof(float), hipMemcpyHostToDevice) );

  // run kernel:
  hypotKernel <<< len/128 + 1, 128 >>> (A_G, B_G, C_G, len);
  getLastCudaError("Kernel execution failed (hypotKernel)");

  // copy results back to CPU:
  checkCudaErrors( hipMemcpy(C, C_G, len*sizeof(float), hipMemcpyDeviceToHost) );

  // Clean up:
  checkCudaErrors( hipFree(A_G) );
  checkCudaErrors( hipFree(B_G) );
  checkCudaErrors( hipFree(C_G) );
  hipDeviceReset();

  return EXIT_SUCCESS;
}
